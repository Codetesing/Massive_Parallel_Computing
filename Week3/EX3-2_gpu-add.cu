#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void add_kernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int SIZE = 5;
	const int a[SIZE] = { 1, 2, 3, 4, 5 };
	const int b[SIZE] = { 10, 20, 30, 40, 50 };
	int c[SIZE] = { 0 };

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));
	hipMalloc((void**)&dev_c, SIZE * sizeof(int));

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	add_kernel << <1, SIZE >> > (dev_c, dev_a, dev_b);

	hipDeviceSynchronize();

	hipError_t err = hipPeekAtLastError();

	if (hipSuccess != err)
	{
		printf("CUDA: ERROR: cuda failure \"%s\"\n", hipGetErrorString(err));
		exit(1);
	}
	else
		printf("CUDA: success\n");

	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	for (int i = 0; i < SIZE; i++)
		printf("%d %d %d\n", a[i], b[i], c[i]);

	return 0;
}