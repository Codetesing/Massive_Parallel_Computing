#include "hip/hip_runtime.h"
#include "./common.cpp"

const unsigned SIZE = 1024 * 1024;

__global__ void kernelVecAdd(float* c, const float* a, const float* b, unsigned n)
{
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < n)
		c[i] = a[i] + b[i];
}

int main(void)
{
	float* vecA = new float[SIZE];
	float* vecB = new float[SIZE];
	float* vecC = new float[SIZE];

	float* dev_vecA = nullptr;
	float* dev_vecB = nullptr;
	float* dev_vecC = nullptr;

	srand(0);

	setNormalizedRandomData(vecA, SIZE);
	setNormalizedRandomData(vecB, SIZE);

	hipMalloc((void**)&dev_vecA, SIZE * sizeof(float));
	hipMalloc((void**)&dev_vecB, SIZE * sizeof(float));
	hipMalloc((void**)&dev_vecC, SIZE * sizeof(float));
	CUDA_CHECK_ERROR();

	ELAPSED_TIME_BEGIN(1);
	hipMemcpy(dev_vecA, vecA, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vecB, vecA, SIZE * sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK_ERROR();

	ELAPSED_TIME_BEGIN(0);
	kernelVecAdd << < SIZE / 1024, 1024 >> > (dev_vecC, dev_vecA, dev_vecB, SIZE);
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);

	CUDA_CHECK_ERROR();

	hipMemcpy(vecC, dev_vecC, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	ELAPSED_TIME_END(1);
	CUDA_CHECK_ERROR();

	hipFree(dev_vecA);
	hipFree(dev_vecB);
	hipFree(dev_vecC);
	CUDA_CHECK_ERROR();

	float sumA = getSum(vecA, SIZE);
	float sumB = getSum(vecB, SIZE);
	float sumC = getSum(vecC, SIZE);
	float diff = fabsf(sumC - (sumA + sumB));

	printf("SIZE = %d\n", SIZE);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("sumC = %f\n", sumC);
	printf("diff(sumC, sumA + sumB) = %f\n", diff);
	printf("diff(sumC, sumA + sumB) / SIZE = %f\n", diff / SIZE);

	for (int i = 0; i < 4; i++)
		printf("%d + %d = %d\n", vecA[i], vecB[i], vecC[i]);

	delete[]vecA;
	delete[]vecB;
	delete[]vecC;

	return 0;
}